#include <cstdio>
#include <exception>
#include "cudaUtil.h"

void assertCudaStatus(hipError_t cudaStatus, const char* msg)
{
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, msg);
        throw std::exception(msg);
    }
}

/**
 * @brief Setting device multiple times apparently has no impact, perf or otherwise.
*/
void setDevice(int deviceId)
{
    assertCudaStatus(hipSetDevice(deviceId), "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
}

void printDeviceInfo(int deviceId)
{
    setDevice(deviceId);

    int major, minor;
    assertCudaStatus(hipDeviceGetAttribute(&major, hipDeviceAttribute_t::hipDeviceAttributeComputeCapabilityMajor, deviceId), "Unable to get device compute capability.");
    assertCudaStatus(hipDeviceGetAttribute(&minor, hipDeviceAttribute_t::hipDeviceAttributeComputeCapabilityMinor, deviceId), "Unable to get device compute capability.");

    int threadsPerSm;
    assertCudaStatus(hipDeviceGetAttribute(&threadsPerSm, hipDeviceAttribute_t::hipDeviceAttributeMaxThreadsPerMultiProcessor, deviceId), "Unable to get device capability.");

    int smCount;
    assertCudaStatus(hipDeviceGetAttribute(&smCount, hipDeviceAttribute_t::hipDeviceAttributeMultiprocessorCount, deviceId), "Unable to get device capability.");

    int kernelCount;
    assertCudaStatus(hipDeviceGetAttribute(&kernelCount, hipDeviceAttribute_t::hipDeviceAttributeConcurrentKernels, deviceId), "Unable to get device capability.");

    int clockRate;
    assertCudaStatus(hipDeviceGetAttribute(&clockRate, hipDeviceAttribute_t::hipDeviceAttributeClockRate, deviceId), "Unable to get device capability.");

    fprintf(stderr, "Device %d compute capability: %d.%d\n", deviceId, major, minor);
    fprintf(stderr, "SM count: %d\n", smCount);
    fprintf(stderr, "Concurrent kernels count: %d\n", kernelCount);
    fprintf(stderr, "Clock rate: %d MHz\n", clockRate / 1000);
    fprintf(stderr, "Threads per SM: %d\n", threadsPerSm);
}

/**
* @brief Create a 2-d source texture for uint16 data type.
* @param tex Output.
* @param cuArray The source data (uint16) array.
* @param samplingType Affects texture filter and read modes.
*/
void createSourceTexture16u(hipTextureObject_t& tex, hipArray_t& cuArray, int samplingType)
{
    hipResourceDesc resourceDesc = {};
    resourceDesc.res.array.array = cuArray;
    resourceDesc.resType = hipResourceTypeArray;

    hipTextureDesc textureDesc = {};
    textureDesc.normalizedCoords = false; // normalized is 0 to 1

    if (samplingType == 1)
    {
        // bilinear
        textureDesc.filterMode = hipFilterModeLinear;
        textureDesc.readMode = hipReadModeNormalizedFloat; // have to do this in order to use texture bilinear
    }
    else
    {
        // for nearest neighbor and bicubic we do nearest neighbor (because we will be doing bicubic ourselves)
        textureDesc.filterMode = hipFilterModePoint;
        textureDesc.readMode = hipReadModeElementType; // orig type (uint16) rather than float
    }

    hipTextureAddressMode addressMode = hipAddressModeBorder; // border means return 0 when off-image rather than clamp the coords (or wrap)
    textureDesc.addressMode[0] = addressMode; 
    textureDesc.addressMode[1] = addressMode;

    auto cudaStatus = hipCreateTextureObject(&tex, &resourceDesc, &textureDesc, NULL);
    assertCudaStatus(cudaStatus, "hipCreateTextureObject failed.");
}

/**
* @brief Create a hipArray and cudaTextureObject and copy source data to device.
* You must free the hipArray and cudaTextureObject when done.
* @param width 
* @param height 
* @param psrc Source data.
* @param samplingType Affects how the texture is set up.
* @param srcArray Output.
* @param srcTexture Output.
*/
void setupSrcImageTexture16u(int width, int height, const uint16_t* psrc, int samplingType, hipArray_t& srcArray, hipTextureObject_t& srcTexture)
{
    size_t imageLen = width * height * sizeof(uint16_t);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint16_t>();

    // src image
    auto cudaStatus = hipMallocArray(&srcArray, &channelDesc, width, height, hipArraySurfaceLoadStore);
    assertCudaStatus(cudaStatus, "CUDA malloc array failed.");
    cudaStatus = hipMemcpyToArray(srcArray, 0, 0, psrc, imageLen, hipMemcpyHostToDevice);
    assertCudaStatus(cudaStatus, "CUDA copy src image to array failed.");
    createSourceTexture16u(srcTexture, srcArray, samplingType);
}
